#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <time.h>
#include <sys/time.h>

#include "../matlib/matrix.h"
#include "matrix_multiplication.h"

int main(int argc, char* argv[]) {
	// Size of vectors
	int SIZE = 1024;
	size_t size = SIZE * SIZE * sizeof(int);
	struct timeval start, end;

	// =====================================
	Matrix h_A = getRandomMatrix(SIZE, SIZE);

	Matrix d_A;
	d_A.width = SIZE;
	d_A.height = SIZE;

	// =====================================

	Matrix h_B = getRandomMatrix(SIZE, SIZE);

	Matrix d_B;
	d_B.width = SIZE;
	d_B.height = SIZE;

	// =====================================

	Matrix h_C = mallocMatrix(SIZE, SIZE);

	// Answer from CUDA
	Matrix h_dC = mallocMatrix(SIZE, SIZE);

	// Answer from CPU
	Matrix d_C;
	d_C.width = SIZE;
	d_C.height = SIZE;

	// Perform CPU calculation
	gettimeofday(&start, NULL);
	matMulCPU(h_A, h_B, h_C, SIZE);
	gettimeofday(&end, NULL);

	printf("CPU calculation ended in %ld\n",
			(end.tv_sec - start.tv_sec) * 1000
					+ (end.tv_usec - start.tv_usec) / 1000);

	// Copy host vectors to device
	gettimeofday(&start, NULL);

	hipMalloc(&d_A.elements, size);
	hipMalloc(&d_B.elements, size);
	hipMalloc(&d_C.elements, size);

	hipMemcpy(d_A.elements, h_A.elements, size, hipMemcpyHostToDevice);
	hipMemcpy(d_B.elements, h_B.elements, size, hipMemcpyHostToDevice);

	// Number of threads in each thread block
	dim3 threadsPerBlock(32, 32);

	// Number of thread blocks in grid
	dim3 numBlocks(SIZE / threadsPerBlock.x, SIZE / threadsPerBlock.y);

	// Execute the kernel
	matMulGPU<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, SIZE);
	hipDeviceSynchronize();

	// Copy array back to host
	hipMemcpy(h_dC.elements, d_C.elements, size, hipMemcpyDeviceToHost);

	// Release device memory
	hipFree(d_A.elements);
	hipFree(d_B.elements);
	hipFree(d_C.elements);

	gettimeofday(&end, NULL);

	printf("GPU calculation ended in %ld\n",
			(end.tv_sec - start.tv_sec) * 1000
					+ (end.tv_usec - start.tv_usec) / 1000);

	bool error = false;
	for (int i = 0; i < SIZE; ++i) {
		for (int j = 0; j < SIZE; ++j) {
			if (h_dC.elements[i * SIZE + j] != h_C.elements[i * SIZE + j]) {
				printf("ERROR %d %d\n", h_dC.elements[i * SIZE + j],
						h_C.elements[i * SIZE + j]);
				error = true;
				break;
			}

		}
		if (error) {
			break;
		}
	}

	// Release host memory
	free(h_A.elements);
	free(h_B.elements);
	free(h_C.elements);

	return 0;
}
